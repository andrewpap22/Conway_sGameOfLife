#include "hip/hip_runtime.h"
/* DESCRIPTION: Conway's Game of Life project for UoA dit, September 2020.
 *
 * AUTHOR:      Andrew Pappas, 1115201500201, UoA, dit.
 * DATE:        September 2020.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "times.h"
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 32

// set FILE_INPUT for random input
#define FILE_INPUT 0

//If output is to be written in file set FILE_OUTPUT to 1 and set the correct output file name,
//otherwise set FILE_OUTPUT to write nothing
#define FILENAMEOUT "myouput.txt"
#define FILE_OUTPUT 1

//If starting or final board is to be printed set the flags to 1 from 0 respectively
#define PRINT_STARTING_BOARD 0
#define PRINT_FINAL_BOARD 0


//Copy real rows to "ghost" rows to make thε array cyclic
__global__ void copyRows(int *board, int dimension)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x + 1;

	if (id <= dimension)
	{
		//Copy first real row to bottom ghost row
		board[(dimension+2)*(dimension+1)+id] = board[(dimension+2)+id];
		//Copy last real row to top ghost row
		board[id] = board[(dimension+2)*dimension + id];
	}
}

//Copy real columns to "ghost" columns to make thε array cyclic
__global__ void copyColumns(int *board, int dimension)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id <= dimension+1)
	{
		//Copy first real column to right most ghost column
		board[id*(dimension+2)+dimension+1] = board[id*(dimension+2)+1];
		//Copy last real column to left most ghost column
		board[id*(dimension+2)] = board[id*(dimension+2) + dimension];
	}
}

//Get the game in the next generation
__global__ void nextGen(int *board, int *newBoard, int dimension)
{
	int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
	int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int id = iy * (dimension+2) + ix;

	int neighbors;

	if (iy <= dimension && ix <= dimension) {

	//count neighbors
	neighbors = board[id+(dimension+2)] + board[id-(dimension+2)] + board[id+1] + board[id-1] + board[id+(dimension+3)] + board[id-(dimension+3)] + board[id-(dimension+1)] + board[id+(dimension+1)];

	int value = board[id];
	//Apply the game's rules
	if (value == 1 && neighbors < 2)
		newBoard[id] = 0;
	else if (value == 1 && (neighbors == 2 || neighbors == 3))
		newBoard[id] = 1;
	else if (value == 1 && neighbors > 3)
		newBoard[id] = 0;
	else if (value == 0 && neighbors == 3)
		newBoard[id] = 1;
	else
		newBoard[id] = value;
	}
}

int main(int argc, char* argv[])
{
	int generations, dimension;
	int *board, *gpuBoard, *gpuNewBoard, *gpuTempBoard;
	FILE *fpOUT;
	//The boards that are going to be used are on-dimension boards even thought the game is
	//played in 2d boards. This is happening to make even faster and easier the coding. The formula
	//which is used to present virtually that 1d board as a 2d was found on the internet as a mathematic formula.

	if(argc!=3){
		printf("Correct usage of program is: %s dimensions generations\n",argv[0]);
		exit(-1);
	}

	dimension = atoi(argv[1]);
	generations = atoi(argv[2]);



	printf("Running Conway's Game of life with:\n");
	printf("Dimension = %d\n",dimension);
	printf("Generations = %d\n",generations);

	size_t size = sizeof(int)*(dimension+2)*(dimension+2);

	//Allocate ram memory equal to the size of the board with the "ghost" cells
	board = (int*)malloc(size);

	if(board == NULL){
		printf("Error in ram memory allocation\n");
		exit(-1);
	}

	srand(0);
	int i,j;

	#if FILE_INPUT==0

	//Initialization of the starting board randomly
	for(i = 1; i<=dimension; i++) {
		for(j = 1; j<=dimension; j++) {
			board[i*(dimension+2)+j] = rand() % 2;
		}
	}

	#endif

	#if PRINT_STARTING_BOARD==1

	printf("\nPrinting starting board:\n");
	for(i = 1; i<=dimension; i++) {
		for(j = 1; j<=dimension; j++) {
			printf("%d ", board[i*(dimension+2)+j]);
		}
		printf("\n");
	}
	printf("\n");

	#endif

	//Allocate gpu memory equal to the size of the board with the "ghost" cells twice
	hipMalloc(&gpuBoard, size);//one for the "old" board in each generation transmission
	hipMalloc(&gpuNewBoard, size);//and one for the "new" one

	//Copying the starting board from the ram to the gpu memory
	hipMemcpy(gpuBoard, board, size, hipMemcpyHostToDevice);

	//Defining the grid of blocks and threads
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE,1);
	int linGrid = (int)ceil(dimension/(float)BLOCK_SIZE);
	dim3 boardSize(linGrid,linGrid,1);
	dim3 cblock_size(BLOCK_SIZE,1,1);
	dim3 rows_size((int)ceil(dimension/(float)cblock_size.x),1,1);
	dim3 columns_size((int)ceil((dimension+2)/(float)cblock_size.x),1,1);

 	//Starting the timer
	timestamp start = getTime();

	int k;
	//Loop for every generation
	for(k=0; k<generations; k++){
		//Creating the "ghost" cells by copying the perimeter cells (rows and columns) to them
		copyRows<<<rows_size, cblock_size>>>(gpuBoard, dimension);
		copyColumns<<<columns_size, cblock_size>>>(gpuBoard, dimension);

		//Proceeding to the next generation of the game saving it to the gpuNewBoard
		nextGen<<<boardSize, blockSize>>>(gpuBoard, gpuNewBoard, dimension);

		//Swaping the old with the new board to avoid re-allocing space
		gpuTempBoard = gpuBoard;
		gpuBoard = gpuNewBoard;
		gpuNewBoard = gpuTempBoard;
	}

 	//Copying the final board the gpu memory to ram
	hipMemcpy(board, gpuBoard, size, hipMemcpyDeviceToHost);

	//Finishing the timer
	float elapsedTime = getElapsedTime(start);

	int sum = 0;
	//Adding up the alive
	for (i = 1; i<=dimension; i++) {
		for (j = 1; j<=dimension; j++) {
			sum += board[i*(dimension+2)+j];
		}
	}

	printf("Total game time: %.2f msecs\n", elapsedTime);
	printf("Total finally alive: %d\n", sum);

	#if PRINT_FINAL_BOARD==1

	printf("\nPrinting final board:\n");
	for(i = 1; i<=dimension; i++) {
		for(j = 1; j<=dimension; j++) {
			printf("%d ", board[i*(dimension+2)+j]);
		}
		printf("\n");
	}
	printf("\n");

	#endif

	#if FILE_OUTPUT==1

	fpOUT = fopen(FILENAMEOUT,"w");
	if (fpOUT == NULL)
	{
		printf("Output file didn't open properly.\n");
		exit(-1);
	}

	for(i = 1; i<=dimension; i++) {
		for(j = 1; j<=dimension; j++) {
			fprintf(fpOUT, "%d ", board[i*(dimension+2)+j]);
		}
		fprintf(fpOUT, "\n");
	}

	fclose(fpOUT);

	#endif

	//Deleting the allocated gpu memory
	hipFree(gpuBoard);
	hipFree(gpuNewBoard);

	//Deleting the allocated ram memory
	free(board);

	return 0;
}
